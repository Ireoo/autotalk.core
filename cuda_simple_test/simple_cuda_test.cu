#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA 内核函数，并行将两个数组相加
__global__ void addArrays(int *a, int *b, int *c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// 检查CUDA错误并输出错误信息
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA错误: %s:%d, %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// 获取并打印CUDA设备信息
void printDeviceInfo() {
    int deviceCount = 0;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
    
    if (deviceCount == 0) {
        printf("未检测到支持CUDA的设备！\n");
        exit(EXIT_FAILURE);
    }
    
    printf("检测到 %d 个CUDA设备:\n", deviceCount);
    
    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        CHECK_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, dev));
        
        printf("\n设备 %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA计算能力: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  全局内存大小: %.2f GB\n", deviceProp.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        printf("  SM数量: %d\n", deviceProp.multiProcessorCount);
        printf("  每个SM的最大线程数: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("  每个线程块的最大线程数: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  最大线程块维度: (%d, %d, %d)\n", 
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("  最大网格维度: (%d, %d, %d)\n", 
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("  CUDA驱动程序版本: %d.%d\n", deviceProp.major, deviceProp.minor);
    }
}

int main() {
    // 打印CUDA设备信息
    printDeviceInfo();
    
    // 定义数组大小
    const int arraySize = 1000000;
    const int byteSize = arraySize * sizeof(int);
    
    // 分配主机内存
    int *h_a = (int*)malloc(byteSize);
    int *h_b = (int*)malloc(byteSize);
    int *h_c = (int*)malloc(byteSize);
    
    // 初始化数组
    for (int i = 0; i < arraySize; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // 分配设备内存
    int *d_a, *d_b, *d_c;
    CHECK_CUDA_ERROR(hipMalloc(&d_a, byteSize));
    CHECK_CUDA_ERROR(hipMalloc(&d_b, byteSize));
    CHECK_CUDA_ERROR(hipMalloc(&d_c, byteSize));
    
    // 将数据从主机复制到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, byteSize, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, byteSize, hipMemcpyHostToDevice));
    
    // 设置CUDA内核执行配置
    int blockSize = 256;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
    
    // 计时
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    CHECK_CUDA_ERROR(hipEventRecord(start));
    
    // 执行CUDA内核
    addArrays<<<numBlocks, blockSize>>>(d_a, d_b, d_c, arraySize);
    
    // 检查内核执行错误
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    // 计算经过的时间
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    
    // 结果复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, byteSize, hipMemcpyDeviceToHost));
    
    // 验证结果
    bool success = true;
    for (int i = 0; i < arraySize; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("验证失败: h_c[%d] = %d, 期望值 = %d\n", i, h_c[i], h_a[i] + h_b[i]);
            success = false;
            break;
        }
    }
    
    if (success) {
        printf("\n计算成功! 处理 %d 个元素耗时 %.3f ms\n", arraySize, milliseconds);
    }
    
    // 释放设备内存
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));
    
    // 释放主机内存
    free(h_a);
    free(h_b);
    free(h_c);
    
    // 清理CUDA事件
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    
    return 0;
} 