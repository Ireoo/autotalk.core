#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel function for vector addition on GPU
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    // Print CUDA version information
    int runtimeVersion = 0;
    hipRuntimeGetVersion(&runtimeVersion);
    printf("CUDA Runtime Version: %d\n", runtimeVersion);
    
    // Print GPU device information
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0)
    {
        printf("No CUDA-capable devices found!\n");
        return -1;
    }
    
    printf("Detected %d CUDA devices:\n", deviceCount);
    
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        
        printf("Device %d: \"%s\"\n", i, deviceProp.name);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total Memory: %.2f GB\n", 
               static_cast<float>(deviceProp.totalGlobalMem) / (1024.0f * 1024.0f * 1024.0f));
        printf("  Multiprocessor Count: %d\n", deviceProp.multiProcessorCount);
    }
    
    // Set array size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("\nPerforming vector addition test (%d elements)...\n", numElements);
    
    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host memory\n");
        exit(EXIT_FAILURE);
    }
    
    // Initialize data
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    
    // Allocate device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    
    hipError_t err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory (A): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory (B): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory (C): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copy data from host to device
    printf("Copying data from host to device...\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device (A): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device (B): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel configuration: %d blocks, %d threads per block\n", blocksPerGrid, threadsPerBlock);
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    // Copy result back to host
    printf("Copying result back to host...\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from device to host: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Verify result
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    
    printf("Vector addition test successful!\n");
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    
    printf("CUDA test completed, all tests passed!\n");
    return 0;
} 